#include "hip/hip_runtime.h"
#include "cdga/Demo.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace cdga {

Demo::Demo() : count(0) { }

Demo::~Demo() { }

__global__ void demoKernel(unsigned int count) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int total = blockDim.x * gridDim.x;
    if(idx == 0) {
        printf("Thread %u of %u: count = %u\n", idx, total, count);
    }
}


void Demo::demo() {
    this->count++;

    // Initialise a runtime cuda context on the default device.
    hipFree(0);

    // Get the current cuda context using the driver api, just to make use of the driver api for testing reasons.

    hipError_t driverAPIStatus;
    hipCtx_t ctx;
    driverAPIStatus = hipCtxGetCurrent(&ctx);
    if (driverAPIStatus != hipSuccess) {
        const char * errstr;
        hipDrvGetErrorString(driverAPIStatus, &errstr);
        fprintf(stderr, "Error: Cuda driver Error %s at %s::%d\n", errstr, __FILE__, __LINE__);
    } else {
        printf("hipCtxGetCurrent success\n");
    }
    
    demoKernel<<<1, 1>>>(this->count);
    hipError_t status;
    hipDeviceSynchronize();
    status = hipGetLastError();
    if (status != hipSuccess) {
        fprintf(stderr, "Error: Cuda Error %s at %s::%d\n", hipGetErrorString(status), __FILE__, __LINE__);
    }

}

}  // namespace cdga

