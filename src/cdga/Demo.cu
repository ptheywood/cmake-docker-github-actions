#include "hip/hip_runtime.h"
#include "cdga/Demo.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Include the class which should be in detail, which interacts with dlopen.
// @todo - might need to make this include conditional?
// @todo - should be in detail
#include "cdga/detail/DSOStuff.h"

namespace cdga {



Demo::Demo() : count(0) { }

Demo::~Demo() { }

__global__ void demoKernel(unsigned int count) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int total = blockDim.x * gridDim.x;
    if(idx == 0) {
        printf("Thread %u of %u: count = %u\n", idx, total, count);
    }
}


void Demo::demo() {


    // dlopen testing. This would want separating / doing much nicer.
    #if defined(USE_DLOPEN_CUDA)

    void * libcuda_handle = detail::DSOStuff::OpenLibraryHandle("libcuda.so.1");
    if (libcuda_handle == nullptr) {
        fprintf(stderr, "Bad stuff happened. @todo \n");
        exit(EXIT_FAILURE);
    }
    // Load required libcuda.so methods. If any fail to load, this will exit, so no need to check the result with the current implementation
    
    hipError_t (*hipCtxGetCurrent)( hipCtx_t* );
    hipError_t (*hipDrvGetErrorString)( hipError_t, const char** );

    hipCtxGetCurrent = (hipError_t (*)( hipCtx_t* )) detail::DSOStuff::SymbolFromLibrary(libcuda_handle, "hipCtxGetCurrent");
    hipDrvGetErrorString = (hipError_t (*)( hipError_t, const char** )) detail::DSOStuff::SymbolFromLibrary(libcuda_handle, "hipDrvGetErrorString");
    #endif
    this->count++;

    // Initialise a runtime cuda context on the default device.
    hipFree(0);

    // Get the current cuda context using the driver api, just to make use of the driver api for testing reasons.

    hipError_t driverAPIStatus;
    hipCtx_t ctx;
    driverAPIStatus = hipCtxGetCurrent(&ctx);
    if (driverAPIStatus != hipSuccess) {
        const char * errstr;
        hipDrvGetErrorString(driverAPIStatus, &errstr);
        fprintf(stderr, "Error: Cuda driver Error %s at %s::%d\n", errstr, __FILE__, __LINE__);
    } else {
        printf("hipCtxGetCurrent success\n");
    }
    
    demoKernel<<<1, 1>>>(this->count);
    hipError_t status;
    hipDeviceSynchronize();
    status = hipGetLastError();
    if (status != hipSuccess) {
        fprintf(stderr, "Error: Cuda Error %s at %s::%d\n", hipGetErrorString(status), __FILE__, __LINE__);
    }

    // @todo move this and do it nicer. 
    #if defined(USE_DLOPEN_CUDA)
        // Close the handle.
        if(libcuda_handle) {
            detail::DSOStuff::CloseLibraryHandle(&libcuda_handle);
        }
    #endif

}

}  // namespace cdga

