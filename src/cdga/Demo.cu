#include "hip/hip_runtime.h"
#include "cdga/Demo.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// If Using dlopen, include the appropraite header. This is only currently implemented for linux.
#if defined(USE_DLOPEN_CUDA) !defined(_MSC_VER)
    #include <dlfcn.h>
#else 
#include <hip/hip_runtime.h>
#endif // USE_DLOPEN

namespace cdga {



Demo::Demo() : count(0) { }

Demo::~Demo() { }

__global__ void demoKernel(unsigned int count) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int total = blockDim.x * gridDim.x;
    if(idx == 0) {
        printf("Thread %u of %u: count = %u\n", idx, total, count);
    }
}


void Demo::demo() {


    // dlopen testing. This would want separating / doing much nicer.
    #if defined(USE_DLOPEN_CUDA)

    // Void pointer to store the dlopen handle
    void * libcuda_handle = nullptr;
    // dlopen error codes.
    char * libcuda_error;
    // Stub for method protos. Do this in a separate file to be included?. dysyms are stored in these?
    // Might need to be per version of the .so (i.e. CUDA 11.2 might ahve different protos that need to be stubbed thatn 110)
    // @todo - could these be std::functions?
    hipError_t (*hipCtxGetCurrent)( hipCtx_t* );
    hipError_t (*hipDrvGetErrorString)( hipError_t, const char** );

    // Open the .so. 
    // @todo - better name calculation.
    libcuda_handle = dlopen("libcuda.so.1", RTLD_LAZY);
    printf("handle opened? %p\n", libcuda_handle);
    if (!libcuda_handle) {
        fprintf(stderr, "dlopen error libcuda_handle: %s\n", dlerror());
        exit(EXIT_FAILURE);
    }
    // Not sure why this is being called again? reset the error incase the handle was oipened perhaps?
    dlerror();
    // Load the symbols into the function pointers.
    hipCtxGetCurrent = (hipError_t (*)( hipCtx_t* )) dlsym(libcuda_handle, "hipCtxGetCurrent");
    libcuda_error = dlerror();
    if (libcuda_error != NULL) {
        fprintf(stderr, "libcuda_error: %s\n", libcuda_error);
        exit(EXIT_FAILURE);
    }
    hipDrvGetErrorString = (hipError_t (*)( hipError_t, const char** )) dlsym(libcuda_handle, "hipDrvGetErrorString");
    libcuda_error = dlerror();
    if (libcuda_error != NULL) {
        fprintf(stderr, "libcuda_error: %s\n", libcuda_error);
        exit(EXIT_FAILURE);
    }
    #endif
    this->count++;

    // Initialise a runtime cuda context on the default device.
    hipFree(0);

    // Get the current cuda context using the driver api, just to make use of the driver api for testing reasons.

    hipError_t driverAPIStatus;
    hipCtx_t ctx;
    driverAPIStatus = hipCtxGetCurrent(&ctx);
    if (driverAPIStatus != hipSuccess) {
        const char * errstr;
        hipDrvGetErrorString(driverAPIStatus, &errstr);
        fprintf(stderr, "Error: Cuda driver Error %s at %s::%d\n", errstr, __FILE__, __LINE__);
    } else {
        printf("hipCtxGetCurrent success\n");
    }
    
    demoKernel<<<1, 1>>>(this->count);
    hipError_t status;
    hipDeviceSynchronize();
    status = hipGetLastError();
    if (status != hipSuccess) {
        fprintf(stderr, "Error: Cuda Error %s at %s::%d\n", hipGetErrorString(status), __FILE__, __LINE__);
    }

    // @todo move this and do it nicer. 
    #if defined(USE_DLOPEN_CUDA)
        // Close the handle.
        if(libcuda_handle) { 
            dlclose(libcuda_handle);
            libcuda_handle = nullptr;
            // @todo - probabyl check for errors again.
        }
    #endif

}

}  // namespace cdga

